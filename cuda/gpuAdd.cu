
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


__global__ void gpuAdd(int nElements, float* x, float* y) {
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < nElements; i += stride)
        y[i] = y[i] + x[i];
}

int main(int argc, char** argv) {
    constexpr int nElements = 1000000; // 1 Million
    float *x;
    hipMallocManaged(&x, nElements * sizeof(float));
    float *y;
    hipMallocManaged(&y, nElements * sizeof(float));
    for (int i = 0; i < nElements; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize = 256;
    int numBlocks = (nElements + blockSize - 1) / blockSize;
    gpuAdd<<<numBlocks, blockSize>>>(nElements, x, y);

    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < nElements; i++)
        maxError = std::fmax(maxError, std::fabs(y[i] - 3.0f));

    std::cout << "Max error:" << maxError << std::endl;
    
    hipFree(x);
    hipFree(y);
    return 0;
}